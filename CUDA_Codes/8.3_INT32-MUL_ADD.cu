#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to perform both INT32 multiplication and addition, and print SM and warp IDs
__global__ void int32Kernel(int *a, int *b, int *c, int *d, int n) {
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    int smId;
    int warpId;

    // Using asm intrinsic to get the SM ID
    asm("mov.u32 %0, %%smid;" : "=r"(smId));
    // Calculate the warp ID
    warpId = globalThreadId / warpSize;

    if (globalThreadId < n) {
        // Perform multiplication
        int mul_result = a[globalThreadId] * b[globalThreadId];
        // Perform addition
        d[globalThreadId] = mul_result + c[globalThreadId];

        // Print SM ID and warp ID
        printf("Thread %d on SM %d in warp %d\n", globalThreadId, smId, warpId);
    }
}

void detectGPUs() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices detected.\n");
        return;
    }

    printf("Detected %d CUDA device(s):\n", deviceCount);
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("\nDevice %d: %s\n", i, deviceProp.name);
        printf("  Total Global Memory: %.2f GB\n", (float)deviceProp.totalGlobalMem / (1024 * 1024 * 1024));
        printf("  Number of SMs: %d\n", deviceProp.multiProcessorCount);
        printf("  Warp Size: %d\n", deviceProp.warpSize);
    }
}

int main() {
    detectGPUs(); // Detect and print GPU properties

    const int N = 128*40*100000;  // Total number of elements for INT32
    const int size_int32 = N * sizeof(int);
    
    // Print sizes of INT32 variables
    printf("Size of int (INT32): %zu bytes\n", sizeof(int));

    // Host memory allocation
    int *h_a_int32, *h_b_int32, *h_c_int32, *h_d_int32;
    h_a_int32 = (int*)malloc(size_int32);
    h_b_int32 = (int*)malloc(size_int32);
    h_c_int32 = (int*)malloc(size_int32);
    h_d_int32 = (int*)malloc(size_int32);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a_int32[i] = i;
        h_b_int32[i] = 2 * i;
        h_c_int32[i] = 3 * i;
    }

    // Device memory allocation
    int *d_a_int32, *d_b_int32, *d_c_int32, *d_d_int32;
    hipMalloc((void**)&d_a_int32, size_int32);
    hipMalloc((void**)&d_b_int32, size_int32);
    hipMalloc((void**)&d_c_int32, size_int32);
    hipMalloc((void**)&d_d_int32, size_int32);

    // Copy data from host to device
    hipMemcpy(d_a_int32, h_a_int32, size_int32, hipMemcpyHostToDevice);
    hipMemcpy(d_b_int32, h_b_int32, size_int32, hipMemcpyHostToDevice);
    hipMemcpy(d_c_int32, h_c_int32, size_int32, hipMemcpyHostToDevice);

    // Time measurement for INT32 operations
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int32Kernel<<<1, 128>>>(d_a_int32, d_b_int32, d_c_int32, d_d_int32, N); // Combined INT32 operations
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("INT32 Multiplication and Addition Time: %.2f ms\n", elapsedTime);

    // Copy results back to host
    hipMemcpy(h_d_int32, d_d_int32, size_int32, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a_int32);
    hipFree(d_b_int32);
    hipFree(d_c_int32);
    hipFree(d_d_int32);

    // Free host memory
    free(h_a_int32);
    free(h_b_int32);
    free(h_c_int32);
    free(h_d_int32);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
