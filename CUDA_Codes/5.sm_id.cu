
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;}

static __device__ __inline__ uint32_t __mylaneid(){
  uint32_t laneid;
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));
  return laneid;}


__global__ void mykernel(){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  printf("I am thread %d | my SM ID is %d | my warp ID is %d | and my warp lane is %d\n", idx, __mysmid(), __mywarpid(), __mylaneid());
}

int main(){

  mykernel<<<41,2>>>();
  hipDeviceSynchronize();
  return 0;
}
