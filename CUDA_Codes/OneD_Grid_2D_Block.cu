#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

__global__ void hello() {

    int threadID = (gridDim.x * blockDim.x * threadIdx.y)+(blockDim.x * blockIdx.x)+threadIdx.x;
    printf("Global TID : %d |I am thread (X : %d, Y: %d, Z: %d) of block (X: %d, Y: %d, Z: %d) in the grid\n",
           threadID,threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z );

}


void printDims(dim3 gridDim, dim3 blockDim) {
    printf("Grid Dimensions : {%d, %d, %d} blocks. \n",
    gridDim.x, gridDim.y, gridDim.z);

    printf("Block Dimensions : {%d, %d, %d} threads.\n",
    blockDim.x, blockDim.y, blockDim.z);
}

int main(int argc, char **argv) {


    dim3 gridDim(2);     // 2 blocks in x direction, y, z default to 1
    dim3 blockDim(2,2);  // 4 threads per block: 2 in x direction, 2 in y

    printDims(gridDim, blockDim);

    printf("From each thread:\n");
    hello<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();      // need for printfs in kernel to flush

    return 0;
}
