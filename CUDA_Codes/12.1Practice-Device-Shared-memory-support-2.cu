#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testDynamicSharedMemory() {
    extern __shared__ int dynamicSharedMemory[];
    int tid = threadIdx.x;
    dynamicSharedMemory[tid] = tid;

    __syncthreads();

    if (tid == 0) {
        int sum = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += dynamicSharedMemory[i];
        }
        printf("Sum: %d\n", sum);
    }
}

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Total shared memory per block: %zu bytes\n", prop.sharedMemPerBlock);

    size_t maxSharedMemory = prop.sharedMemPerBlock;
    int threadsPerBlock = 256;
    int numBlocks = 4;

    // Launch the kernel with the maximum dynamic shared memory allocation
    testDynamicSharedMemory<<<numBlocks, threadsPerBlock, maxSharedMemory>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    return 0;
}
